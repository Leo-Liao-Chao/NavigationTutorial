#include <stdio.h>
#include "myutil.h"

using namespace std;

void GetCudaDeviceCount( int &num_gpus )
{
    hipGetDeviceCount( &num_gpus );

    for ( int i = 0; i < num_gpus; ++ i )
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        printf("   %d: %s\n", i, dprop.name);
    }
}